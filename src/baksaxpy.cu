#include "hip/hip_runtime.h"
/**
 * Compute saxpy
 * - on CPU : serial and OpenMP version
 * - on GPU : first using CUDA, then library CuBLAS
 *
 * compare timings.
 *
 */



// =========================
// standard imports
// =========================
#include <stdio.h>
#include <stdlib.h>
#include <iostream>


#include <iostream>
#include <fstream>
#include <sstream>
#include <vector>
#include <map>
#include <unordered_map>

#include "cutlass/cutlass.h"
#include "cutlass/gemm/gemm.h"
#include "cutlass/gemm/kernel/gemm_grouped.h"
#include "cutlass/gemm/kernel/default_gemm_grouped.h"
#include "cutlass/gemm/device/gemm_grouped.h"
#include "cutlass/gemm/device/gemm_universal.h"

#include "cutlass/util/command_line.h"
#include "cutlass/util/distribution.h"
#include "cutlass/util/device_memory.h"
#include "cutlass/util/tensor_view_io.h"
#include "cutlass/util/host_tensor.h"
#include "cutlass/util/reference/host/gemm_complex.h"
#include "cutlass/util/reference/device/gemm_complex.h"
#include "cutlass/util/reference/host/tensor_compare.h"
#include "cutlass/util/reference/host/tensor_copy.h"
#include "cutlass/util/reference/device/tensor_fill.h"
#include "cutlass/util/reference/host/tensor_norm.h"

#include "cutlass/layout/matrix.h"
#include "cutlass/gemm/kernel/gemm_grouped.h"
#include "cutlass/gemm/kernel/gemm_transpose_operands.h"
#include "cutlass/gemm/kernel/default_gemm.h"
#include "cutlass/gemm/kernel/default_gemm_complex.h"
#include "cutlass/gemm/device/default_gemm_configuration.h"
#include "cutlass/gemm/gemm.h"

#include "cutlass/epilogue/threadblock/epilogue_with_visitor.h"
#include "cutlass/fast_math.h"

// =========================
// CUDA imports
// =========================
#include <hip/hip_runtime.h>
#include <hipblas.h>

#include <cutlass/cutlass.h>
#include <cutlass/numeric_types.h>
#include <cutlass/core_io.h>
#include <cutlass/array.h>
#include <cutlass/gemm/device/gemm.h>
#include <cutlass/util/host_tensor.h>

#include <cutlass/layout/matrix.h>
#include <cutlass/util/reference/host/tensor_fill.h>
#include <cutlass/util/reference/device/tensor_fill.h>
#include <cutlass/util/host_tensor.h>
#include <cutlass/util/tensor_view_io.h>

#include <cutlass/numeric_types.h>
#include <cutlass/layout/matrix.h>

#include <cutlass/util/host_tensor.h>
#include <cutlass/util/reference/host/gemm.h>
#include "cutlass/cutlass.h"
#include "cutlass/gemm/gemm.h"
#include "cutlass/gemm/device/gemm_grouped.h"

#include "cutlass/util/command_line.h"
#include "cutlass/util/distribution.h"
#include "cutlass/util/device_memory.h"
#include "cutlass/util/tensor_view_io.h"
#include "cutlass/util/host_tensor.h"
#include "cutlass/util/reference/host/gemm_complex.h"
#include "cutlass/util/reference/device/gemm_complex.h"
#include "cutlass/util/reference/host/tensor_compare.h"
#include "cutlass/util/reference/host/tensor_copy.h"
#include "cutlass/util/reference/device/tensor_fill.h"
#include "cutlass/util/reference/host/tensor_norm.h"

#include "cutlass/layout/matrix.h"
#include "cutlass/gemm/kernel/default_gemm.h"
#include "cutlass/gemm/kernel/default_gemm_complex.h"
#include "cutlass/gemm/device/default_gemm_configuration.h"
#include "cutlass/gemm/gemm.h"

#include "cutlass/fast_math.h"

#include "default_fmha_grouped.h"
#include "tzq_cuda.h"

#include "flash.h"
#include "kernel_traits.h"
#include "flash_fwd_kernel.h"
#include "flash_fwd_launch_template.h"
#include "static_switch.h"

#include "flash_api.cu"

//#include <Python.h>
//#include <torch/extension.h>
//#include "add2.h"

//Flash_fwd_params ffp;

struct Atten_params{
  Flash_fwd_params ffp;
  int batch_size;
  int seq_length;
  int seq_length_kv;

  
};

hipStream_t stream;

Atten_params  atten_params;

/// Result structure
struct Result {

  double runtime_ms;
  double gflops;
  cutlass::Status status;
  hipError_t error;
  bool passed;

  //
  // Methods
  //

  Result(
    double runtime_ms = 0,
    double gflops = 0,
    cutlass::Status status = cutlass::Status::kSuccess,
    hipError_t error = hipSuccess
  ):
    runtime_ms(runtime_ms), gflops(gflops), status(status), error(error), passed(true) { }
};
Result result;

// Command line options parsing
struct Options {

  bool help;
  bool error;
  bool reference_check;
  bool use_mask;
  bool causal;
  bool fixed_seq_length;

  std::vector<cutlass::gemm::GemmCoord> problem_sizes0;
  std::vector<cutlass::gemm::GemmCoord> problem_sizes1;

  std::vector<cutlass::gemm::GemmCoord> problem_sizes0_real;
  std::vector<cutlass::gemm::GemmCoord> problem_sizes1_real;

  int alignment;
  int head_number;
  int batch_size;
  int head_size;
  int head_size_v;
  int seq_length;
  int seq_length_kv;
  int iterations;
  int problem_count;

  // alpha0, alpha1 and beta are fixed 
  // in this multi-head attention example
  float alpha0;
  float alpha1;
  float beta;

  cutlass::gemm::kernel::GroupScheduleMode scheduler_mode;

  //
  // Methods
  // 

  Options():
    help(false),
    error(false),
    alignment(1),
    reference_check(true),
    head_number(12),
    batch_size(16),
    head_size(64),
    head_size_v(64),
    seq_length(1024),
    seq_length_kv(1024),
    use_mask(false),
    iterations(20),
    causal(false),
    fixed_seq_length(false),
    problem_count(batch_size * head_number),
    scheduler_mode(cutlass::gemm::kernel::GroupScheduleMode::kDeviceOnly)
  { }

  // Parses the command line
  void parse(int argc, char const **args) {
    cutlass::CommandLine cmd(argc, args);

    if (cmd.check_cmd_line_flag("help")) {
      help = true;
      return;
    }

    cmd.get_cmd_line_argument("alignment", alignment, 1);
    cmd.get_cmd_line_argument("head_number", head_number, 12);
    cmd.get_cmd_line_argument("batch_size", batch_size, 16);
    cmd.get_cmd_line_argument("head_size", head_size, 64);
    cmd.get_cmd_line_argument("head_size_v", head_size_v, head_size);
    cmd.get_cmd_line_argument("seq_length", seq_length, 1024);
    cmd.get_cmd_line_argument("seq_length_kv", seq_length_kv, seq_length);
    cmd.get_cmd_line_argument("use_mask", use_mask, false);
    cmd.get_cmd_line_argument("iterations", iterations, 20);
    cmd.get_cmd_line_argument("reference-check", reference_check, true);
    cmd.get_cmd_line_argument("causal", causal, true);
    cmd.get_cmd_line_argument("fixed_seq_length", fixed_seq_length, false);

    std::vector<std::string> scheduler_mode_strs;
    cmd.get_cmd_line_arguments("scheduler-mode", scheduler_mode_strs);

    atten_params.ffp.d = head_size;
    atten_params.ffp.h = head_number;
    atten_params.batch_size = batch_size;
    atten_params.seq_length = seq_length;
    atten_params.seq_length_kv = seq_length_kv;

    if (!scheduler_mode_strs.empty()) {
      if (scheduler_mode_strs.size() > 1) {
        std::cerr << "Only one scheduler mode may be passed in" << std::endl;
        error = true;
        return;
      }
      std::string scheduler_mode_str = scheduler_mode_strs[0];
      if (scheduler_mode_str == "kDeviceOnly") {
        scheduler_mode = cutlass::gemm::kernel::GroupScheduleMode::kDeviceOnly;
      } else if (scheduler_mode_str == "kHostPrecompute") {
        scheduler_mode = cutlass::gemm::kernel::GroupScheduleMode::kHostPrecompute;
      } else {
          std::cerr << "Unrecognized scheduler mode '" << scheduler_mode_str << "'" << std::endl;
          error = true;
          return;
      }
    }

    randomize_problems();
  }

  void randomize_problems() {

    problem_count = head_number * batch_size;

    problem_sizes0.reserve(problem_count);
    problem_sizes1.reserve(problem_count);

    // When using mask, the original inputs are not padded
    // and we need to save these info.
    if (use_mask) {
      problem_sizes0_real.reserve(problem_count);
      problem_sizes1_real.reserve(problem_count);
    }

    for (int i = 0; i < batch_size; ++i) { // loop over batch size
      // problems belonging to the same batch share the same seq len

      int m_real, mkv_real;
      if (fixed_seq_length) {
        m_real = seq_length;
        mkv_real = seq_length_kv;
      } else {
        m_real = (rand() % seq_length) + 1;

        // Only randomize seq_length_kv if it was set to a different value than
        // seq_length originally.
        if (seq_length != seq_length_kv) {
          mkv_real = (rand() % seq_length_kv) + 1;
        } else {
          mkv_real = m_real;
        }
      }

      int m = (m_real + alignment - 1) / alignment * alignment;
      int mkv = (mkv_real + alignment - 1) / alignment * alignment;
      int k0 = head_size;
      int k1 = head_size_v;

      for (int j = 0; j < head_number; ++j) { // loop over head number
        cutlass::gemm::GemmCoord problem0(m, mkv, k0);
        cutlass::gemm::GemmCoord problem1(m, k1, mkv);

        problem_sizes0.push_back(problem0);
        problem_sizes1.push_back(problem1);

        if (use_mask) {
          cutlass::gemm::GemmCoord problem0_real(m_real, mkv_real, k0);
          cutlass::gemm::GemmCoord problem1_real(m_real, k1, mkv_real);
          problem_sizes0_real.push_back(problem0_real);
          problem_sizes1_real.push_back(problem1_real);
        }

      }
    }
  }

  void print_problems() {
    std::cout << "     Running " << batch_size << " batches, each with " << head_number << " heads of size " << head_size << ":" << std::endl;
    for (int i = 0; i < batch_size; ++i) {
      int idx = i * head_number;
      std::cout << "       [" << i << "] seq_length = " << problem_sizes0[idx].m() << " seq_length_kv = " << problem_sizes0[idx].n() << std::endl;
    }
  }


  /// Compute performance in GFLOP/s
  double gflops(double runtime_s) const {

    // Number of real-valued multiply-adds 
    int64_t fops = int64_t();

    for (int i = 0; i < problem_sizes0.size(); ++i) {
      auto const& problem0 = problem_sizes0[i];
      auto const& problem1 = problem_sizes1[i];

      for (int row = 0; row < problem0.m(); ++row) {
        int num_cols0 = problem0.n();
        if (causal) {
          num_cols0 = std::min(row + 1, num_cols0);
        }
        // P <- Q . K_t
        fops += 2 * num_cols0 * problem0.k();
        // P <- exp(P - max(P))
        fops += 2 * num_cols0;
        // S <- sum(P)
        fops += num_cols0 - 1;
        // O <- P . V
        fops += 2 * num_cols0 * problem1.n();
        // O <- O / S
        fops += num_cols0 * problem1.n();
      }
    }

    return double(fops) / double(1.0e9) / runtime_s;
  }
};


template <typename Attention>
class TestbedAttention {
public:

  //
  // Type definitions
  //

  using scalar_t = typename Attention::GemmKernel::scalar_t;
  using accum_t = typename Attention::GemmKernel::accum_t;
  using output_t = typename Attention::GemmKernel::output_t;
  using output_accum_t = typename Attention::GemmKernel::output_accum_t;

  using ElementQ = scalar_t;
  using ElementK = scalar_t;
  using ElementP = accum_t;
  using ElementAccumulator = accum_t;
  using ElementV = scalar_t;
  using ElementO = output_t;
  using ElementOAccum = output_accum_t;

  using ElementCompute = accum_t;

  using ElementNorm = accum_t;
  using ElementSum = accum_t;
  using ElementSoftmaxCompute = accum_t;

  using LayoutQ = cutlass::layout::RowMajor;
  using LayoutK = cutlass::layout::ColumnMajor;
  using LayoutP = cutlass::layout::RowMajor;
  using LayoutV = cutlass::layout::RowMajor;
  using LayoutO = cutlass::layout::RowMajor;

  using MatrixCoord = typename LayoutP::TensorCoord;

  static bool const kNeedsOutputAccumulatorBuffer = Attention::GemmKernel::kNeedsOutputAccumulatorBuffer;

private:

  //
  // Data members
  //

  Options & options;

  /// Initialization
  cutlass::Distribution::Kind init_Q;
  cutlass::Distribution::Kind init_K;
  cutlass::Distribution::Kind init_P;
  cutlass::Distribution::Kind init_V;
  cutlass::Distribution::Kind init_O;
  uint32_t seed;

  cutlass::DeviceAllocation<cutlass::gemm::GemmCoord> problem_sizes_device0;
  cutlass::DeviceAllocation<cutlass::gemm::GemmCoord> problem_sizes_device1;
  cutlass::DeviceAllocation<cutlass::gemm::GemmCoord> problem_sizes_device0_real;

  std::vector<int64_t> offset_Q;
  std::vector<int64_t> offset_K;
  std::vector<int64_t> offset_P;
  std::vector<int64_t> offset_V;
  std::vector<int64_t> offset_O;

  std::vector<int64_t> ldq_host;
  std::vector<int64_t> ldk_host;
  std::vector<int64_t> ldp_host;
  std::vector<int64_t> ldv_host;
  std::vector<int64_t> ldo_host;
  std::vector<int64_t> seqlen_host;

  cutlass::DeviceAllocation<int64_t> ldq;
  cutlass::DeviceAllocation<int64_t> ldk;
  cutlass::DeviceAllocation<int64_t> ldp;
  cutlass::DeviceAllocation<int64_t> ldv;
  cutlass::DeviceAllocation<int64_t> ldo;
  cutlass::DeviceAllocation<int64_t> seqlen;

  cutlass::DeviceAllocation<ElementQ> block_Q;
  cutlass::DeviceAllocation<ElementK> block_K;
  cutlass::DeviceAllocation<ElementP> block_P;
  cutlass::DeviceAllocation<ElementV> block_V;
  cutlass::DeviceAllocation<ElementO> block_O;
  cutlass::DeviceAllocation<ElementOAccum> block_O_accumulate;
  cutlass::DeviceAllocation<ElementNorm> block_Norm;
  cutlass::DeviceAllocation<ElementSum> block_Sum;

  cutlass::DeviceAllocation<int64_t> offset_P_Device;

  cutlass::DeviceAllocation<ElementQ *> ptr_Q;
  cutlass::DeviceAllocation<ElementK *> ptr_K;
  cutlass::DeviceAllocation<ElementP *> ptr_P;
  cutlass::DeviceAllocation<ElementV *> ptr_V;
  cutlass::DeviceAllocation<ElementO *> ptr_O;
  cutlass::DeviceAllocation<ElementOAccum *> ptr_O_accumulate;


public:

  //
  // Methods
  //

  TestbedAttention(
    Options &options_,
    cutlass::Distribution::Kind init_Q_ = cutlass::Distribution::Uniform,
    cutlass::Distribution::Kind init_K_ = cutlass::Distribution::Uniform,
    cutlass::Distribution::Kind init_P_ = cutlass::Distribution::Uniform,
    cutlass::Distribution::Kind init_V_ = cutlass::Distribution::Uniform,
    cutlass::Distribution::Kind init_O_ = cutlass::Distribution::Uniform,
    uint32_t seed_ = 3080
  ):
    options(options_), init_Q(init_Q_), init_K(init_K_), init_P(init_P_), init_V(init_V_), init_O(init_O_), seed(seed_) { }

  int problem_count() const {
    return (options.head_number * options.batch_size);
  }

private:

  /// Helper to initialize a tensor view
  template <typename Element>
  void initialize_tensor_(
    Element *ptr,
    size_t capacity, 
    cutlass::Distribution::Kind dist_kind,
    uint32_t seed) {

    if (dist_kind == cutlass::Distribution::Uniform) {

      Element scope_max, scope_min;
      int bits_input = cutlass::sizeof_bits<Element>::value;
      int bits_output = cutlass::sizeof_bits<ElementP>::value;

      if (bits_input == 1) {
        scope_max = 2;
        scope_min = 0;
      } else if (bits_input <= 8) {
        scope_max = 2;
        scope_min = -2;
      } else if (bits_output == 16) {
        scope_max = 8;
        scope_min = -8;
      } else {
        scope_max = 8;
        scope_min = -8;
      }

      cutlass::reference::device::BlockFillRandomUniform(
        ptr, capacity, seed, scope_max, scope_min, 0);
    } 
    else if (dist_kind == cutlass::Distribution::Gaussian) {

      cutlass::reference::device::BlockFillRandomGaussian(
        ptr, capacity, seed, Element(), Element(0.5f));
    }
    else if (dist_kind == cutlass::Distribution::Sequential) {

      // Fill with increasing elements
      cutlass::reference::device::BlockFillSequential(
        ptr, capacity, Element(1), Element());
    } 
    else {

      // Fill with all 1s
      cutlass::reference::device::BlockFillSequential(
        ptr, capacity, Element(), Element(1));
    }
  }

  /// Initializes data structures
  void initialize_() {

    //
    // Set scalors for the mha example
    //

    options.alpha0 = 1.0f / sqrt(float(options.head_size));
    options.alpha1 = 1.0f;
    options.beta = 0;

    //
    // Choose random problem sizes
    //

    // construct a few problems of random sizes
    srand(seed);

    int64_t total_elements_Q = 0;
    int64_t total_elements_K = 0;
    int64_t total_elements_P = 0;
    int64_t total_elements_V = 0;
    int64_t total_elements_O = 0;

    ldq_host.resize(problem_count());
    ldk_host.resize(problem_count());
    ldp_host.resize(problem_count());
    ldv_host.resize(problem_count());
    ldo_host.resize(problem_count());
    seqlen_host.resize(problem_count());

    for (int32_t i = 0; i < problem_count(); ++i) {

      auto problem0 = options.problem_sizes0.at(i);
      auto problem1 = options.problem_sizes1.at(i);

      ldq_host.at(i) = LayoutQ::packed({problem0.m(), problem0.k()}).stride(0);
      ldk_host.at(i) = LayoutK::packed({problem0.k(), problem0.n()}).stride(0);
      ldp_host.at(i) = LayoutP::packed({problem0.m(), problem0.n()}).stride(0);
      ldv_host.at(i) = LayoutV::packed({problem1.k(), problem1.n()}).stride(0);
      ldo_host.at(i) = LayoutO::packed({problem1.m(), problem1.n()}).stride(0);

      // m = n for attention problems.
      seqlen_host.at(i) = problem0.m();

      offset_Q.push_back(total_elements_Q);
      offset_K.push_back(total_elements_K);
      offset_P.push_back(total_elements_P);
      offset_V.push_back(total_elements_V);
      offset_O.push_back(total_elements_O);

      int64_t elements_Q = problem0.m() * problem0.k();
      int64_t elements_K = problem0.k() * problem0.n();
      int64_t elements_P = problem0.m() * problem0.n();
      int64_t elements_V = problem1.k() * problem1.n();
      int64_t elements_O = problem1.m() * problem1.n();

      total_elements_Q += elements_Q;
      total_elements_K += elements_K;
      total_elements_P += elements_P;
      total_elements_V += elements_V;
      total_elements_O += elements_O;

    }

    problem_sizes_device0.reset(problem_count());
    problem_sizes_device1.reset(problem_count());
    problem_sizes_device0.copy_from_host(options.problem_sizes0.data());
    problem_sizes_device1.copy_from_host(options.problem_sizes1.data());

    if (options.use_mask) {
      problem_sizes_device0_real.reset(problem_count());
      problem_sizes_device0_real.copy_from_host(options.problem_sizes0_real.data());
    }

    ldq.reset(problem_count());
    ldk.reset(problem_count());
    ldp.reset(problem_count());
    ldv.reset(problem_count());
    ldo.reset(problem_count());
    seqlen.reset(problem_count());

    ldq.copy_from_host(ldq_host.data());
    ldk.copy_from_host(ldk_host.data());
    ldp.copy_from_host(ldp_host.data());
    ldv.copy_from_host(ldv_host.data());
    ldo.copy_from_host(ldo_host.data());
    seqlen.copy_from_host(seqlen_host.data());

    //
    // Assign pointers
    //

    block_Q.reset(total_elements_Q);
    block_K.reset(total_elements_K);
    block_P.reset(total_elements_P);
    block_V.reset(total_elements_V);
    block_O.reset(total_elements_O);

    if (kNeedsOutputAccumulatorBuffer) {
      block_O_accumulate.reset(total_elements_O);
    }

    offset_P_Device.reset(problem_count());

    // sync offset with device
    cutlass::device_memory::copy_to_device(offset_P_Device.get(), offset_P.data(), offset_P.size());

    std::vector<ElementQ *> ptr_Q_host(problem_count());
    std::vector<ElementK *> ptr_K_host(problem_count());
    std::vector<ElementP *> ptr_P_host(problem_count());
    std::vector<ElementV *> ptr_V_host(problem_count());
    std::vector<ElementO *> ptr_O_host(problem_count());
    std::vector<ElementOAccum *> ptr_O_accumulate_host(problem_count());
    std::vector<ElementNorm *> ptr_norm_host(problem_count());
    std::vector<ElementSum *> ptr_sum_host(problem_count());

    for (int32_t i = 0; i < problem_count(); ++i) {
      ptr_Q_host.at(i) = block_Q.get() + offset_Q.at(i);
      ptr_K_host.at(i) = block_K.get() + offset_K.at(i);
      ptr_P_host.at(i) = block_P.get() + offset_P.at(i);
      ptr_V_host.at(i) = block_V.get() + offset_V.at(i);
      ptr_O_host.at(i) = block_O.get() + offset_O.at(i);

      if (kNeedsOutputAccumulatorBuffer) {
        ptr_O_accumulate_host.at(i) = block_O_accumulate.get() + offset_O.at(i);
      }
    }

    atten_params.ffp.k_ptr = ptr_K_host.data();
    atten_params.ffp.q_ptr = ptr_Q_host.data();
    atten_params.ffp.v_ptr = ptr_V_host.data();

    ptr_Q.reset(problem_count());
    ptr_Q.copy_from_host(ptr_Q_host.data());
    
    ptr_K.reset(problem_count());
    ptr_K.copy_from_host(ptr_K_host.data());
    
    ptr_P.reset(problem_count());
    ptr_P.copy_from_host(ptr_P_host.data());

    ptr_V.reset(problem_count());
    ptr_V.copy_from_host(ptr_V_host.data());

    ptr_O.reset(problem_count());
    ptr_O.copy_from_host(ptr_O_host.data());

    if (kNeedsOutputAccumulatorBuffer) {
      ptr_O_accumulate.reset(problem_count());
      ptr_O_accumulate.copy_from_host(ptr_O_accumulate_host.data());
    }

    //
    // Initialize the problems of the workspace
    //

    initialize_tensor_(block_Q.get(), total_elements_Q, init_Q, seed + 1);
    initialize_tensor_(block_K.get(), total_elements_K, init_K, seed + 2);
    initialize_tensor_(block_V.get(), total_elements_V, init_V, seed + 3);

  }

  template<typename Element>
  bool verify_tensor_(std::vector<Element> vector_Input, \
                       std::vector<Element> vector_Input_Ref,
                       int64_t verify_length = -1) {

    int64_t size = (vector_Input.size() < vector_Input_Ref.size()) ? vector_Input.size() : vector_Input_Ref.size();
    size = (verify_length == -1) ? size : verify_length;

    // 0.05 for absolute error
    float abs_tol = 5e-2f;
    // 10% for relative error
    float rel_tol = 1e-1f;
    for (int64_t i = 0; i < size; ++i) {
      float diff = (float)(vector_Input.at(i) - vector_Input_Ref.at(i));
      float abs_diff = fabs(diff);
      float abs_ref = fabs((float)vector_Input_Ref.at(i) + 1e-5f);
      float relative_diff = abs_diff / abs_ref;
      if ( (isnan(abs_diff) || isinf(abs_diff)) ||  (abs_diff > abs_tol && relative_diff > rel_tol)) {
        printf("[%d/%d] diff = %f, rel_diff = %f, {computed=%f, ref=%f}.\n", int(i), int(size), abs_diff, relative_diff, (float)(vector_Input.at(i)), (float)(vector_Input_Ref.at(i)));
        return false;
      }

    }
    
    return true;
  }

  /// Verifies the result is a GEMM
  bool verify_() {

    bool passed = true;

    for (int32_t i = 0; i < problem_count(); ++i) {
      cutlass::gemm::GemmCoord problem0 = options.problem_sizes0.at(i);
      cutlass::gemm::GemmCoord problem1 = options.problem_sizes1.at(i);

      LayoutQ layout_Q(ldq_host.at(i));
      LayoutK layout_K(ldk_host.at(i));
      LayoutP layout_P(ldp_host.at(i));
      LayoutV layout_V(ldv_host.at(i));
      LayoutO layout_O(ldo_host.at(i));

      MatrixCoord extent_Q{problem0.m(), problem0.k()};
      MatrixCoord extent_K{problem0.k(), problem0.n()};
      MatrixCoord extent_P{problem0.m(), problem0.n()};
      MatrixCoord extent_V{problem1.k(), problem1.n()};
      MatrixCoord extent_O{problem1.m(), problem1.n()};

      cutlass::TensorView<ElementQ, LayoutQ> view_Q(block_Q.get() + offset_Q.at(i), layout_Q, extent_Q);
      cutlass::TensorView<ElementK, LayoutK> view_K(block_K.get() + offset_K.at(i), layout_K, extent_K);
      cutlass::TensorView<ElementP, LayoutP> view_P(block_P.get() + offset_P.at(i), layout_P, extent_P);
      cutlass::TensorView<ElementV, LayoutV> view_V(block_V.get() + offset_V.at(i), layout_V, extent_V);

      cutlass::DeviceAllocation<ElementP>    block_Ref(layout_P.capacity(extent_P));
      cutlass::TensorView<ElementP, LayoutP> view_Ref_device(block_Ref.get(), layout_P, extent_P);

      cutlass::DeviceAllocation<ElementO>    block_Ref_O(layout_O.capacity(extent_O));
      cutlass::TensorView<ElementO, LayoutO> view_Ref_O_device(block_Ref_O.get(), layout_O, extent_O);
      cutlass::reference::device::TensorFill(view_Ref_O_device, ElementO(0));

      // Reference GEMM
      cutlass::reference::device::GemmComplex<
          ElementQ, LayoutQ,
          ElementK, LayoutK,
          ElementP, LayoutP, 
          ElementCompute, ElementAccumulator
      >(
        problem0,
        ElementAccumulator(options.alpha0), 
        view_Q,
        Attention::GemmKernel::MM0::Mma::kTransformA,
        view_K,
        Attention::GemmKernel::MM0::Mma::kTransformB,
        ElementAccumulator(options.beta), 
        view_P, 
        view_Ref_device, 
        ElementAccumulator(0)
      );

      // Compute softmax for P. We need to explicitly compute softmax
      // over P because softmax is fused to the second GEMM in the
      // profiled implementation.
      std::vector<ElementP> matrix_Ref(layout_P.capacity(extent_P));
      cutlass::device_memory::copy_to_host(matrix_Ref.data(), block_Ref.get(), matrix_Ref.size());
      cutlass::TensorView<ElementP, LayoutP> view_Ref_host(matrix_Ref.data(), layout_P, extent_P);
      std::vector<ElementNorm> vector_Norm_Ref(problem0.m());
      std::vector<ElementSum> vector_Sum_Ref(problem0.m());

      int n_dim = options.use_mask ? options.problem_sizes0_real.at(i).n() : problem0.n();

      // Compute softmax for reference matrix
      for (int m = 0; m < problem0.m(); m++) {
        int n_dim_row = n_dim;
        if (options.causal) {
          n_dim_row = std::min(m + 1, n_dim);
        }
        ElementSoftmaxCompute max = ElementSoftmaxCompute(view_Ref_host.ref().at({m, 0}));
        for (int n = 1; n < n_dim_row; n++) {
           max = std::max(max, ElementSoftmaxCompute(view_Ref_host.ref().at({m, n})));
        }

        vector_Norm_Ref.at(m) = ElementNorm(max);

        ElementSoftmaxCompute sum = ElementSoftmaxCompute();
        for (int n = 0; n < n_dim_row; n++) {
          sum += std::exp( ElementSoftmaxCompute(view_Ref_host.ref().at({m, n})) - max );
        }
        ElementSoftmaxCompute inv_sum = ElementSoftmaxCompute(1.0f / sum);

        vector_Sum_Ref.at(m) = ElementSum(inv_sum);

        for (int n = 0; n < n_dim_row; n++) {
          view_Ref_host.ref().at({m, n}) = ElementP(
            std::exp( ElementSoftmaxCompute(view_Ref_host.ref().at({m, n})) - max ) * inv_sum
          );
        }
        // Mask out the rest of the attention matrix
        for (int n = n_dim_row; n < n_dim; ++n) {
          view_Ref_host.ref().at({m, n}) = ElementP(0);
        }

      }

      // when not using mask, problem_real and problem share the same sizes
      if (options.use_mask) {
        for (int m = 0; m < problem0.m(); m++) {
          for (int n = n_dim; n < problem0.n(); n++) {
            view_Ref_host.ref().at({m, n}) = ElementP(0);
          }
        }
      }

      cutlass::device_memory::copy_to_device(block_P.get() + offset_P.at(i), matrix_Ref.data(), matrix_Ref.size());

      // Reference GEMM
      cutlass::reference::device::GemmComplex<
          ElementP, LayoutP,
          ElementV, LayoutV,
          ElementO, LayoutO, 
          ElementCompute, ElementAccumulator
      >(
        problem1,
        ElementAccumulator(options.alpha1), 
        view_P,
        Attention::GemmKernel::MM0::Mma::kTransformA,
        view_V,
        Attention::GemmKernel::MM0::Mma::kTransformB,
        ElementAccumulator(options.beta), 
        view_Ref_O_device, 
        view_Ref_O_device, 
        ElementAccumulator(0)
      );

      // Copy to host memory
      cutlass::TensorView<ElementP, LayoutP> view_Ref(matrix_Ref.data(), layout_P, extent_P);

      std::vector<ElementO> matrix_O(layout_O.capacity(extent_O));
      cutlass::device_memory::copy_to_host(matrix_O.data(),   block_O.get() + offset_O.at(i), matrix_O.size());
      std::vector<ElementO> matrix_Ref_O(layout_O.capacity(extent_O));
      cutlass::device_memory::copy_to_host(matrix_Ref_O.data(), block_Ref_O.get(), matrix_Ref_O.size());


      bool verified_O = false;
      if (!verified_O) {
        verified_O = verify_tensor_<ElementO>(matrix_O, matrix_Ref_O);
      }

      passed = passed && verified_O;

      if (!passed) {
        std::cerr << "\n***\nError - problem " << i << " failed the QA check\n***\n" << std::endl;

        if (!verified_O) {
          std::cout << "Final matrix output is incorrect" << std::endl;
        }

        return passed;
      }

    }

    return passed;
  }

public:

  Result profile() {

    Result result;
    result.passed = false;

    int threadblock_count = Attention::sufficient(options.problem_sizes1.data(), options.problem_count);

    // Early exit
    if (!threadblock_count) {
      std::cout << "Active CUDA device lacks hardware resources to run CUTLASS Grouped FMHA kernel." << std::endl;
      return result;
    }

    result.passed = false;

    // Initialize the problem
    initialize_();

    typename Attention::Arguments args(
      problem_sizes_device0.get(),
      problem_sizes_device1.get(),
      options.problem_count,
      threadblock_count,
      ptr_Q.get(),
      ptr_K.get(),
      ptr_P.get(),
      ptr_V.get(),
      ptr_O.get(),
      ptr_O_accumulate.get(),
      ldq.get(),
      ldk.get(),
      ldp.get(),
      ldv.get(),
      ldo.get(),
      options.causal,
      options.alpha0,
      options.problem_sizes1.data()
    );

    Attention fmha;

    size_t workspace_size = fmha.get_workspace_size(args);
    cutlass::DeviceAllocation<uint8_t> workspace(workspace_size);

    result.status = fmha.initialize(args, workspace.get());

    if (result.status != cutlass::Status::kSuccess) {
      std::cerr << "Failed to initialize CUTLASS Grouped FMHA kernel." << std::endl;
      return result;
    }

    // Run the grouped FMHA object
    result.status = fmha.run();

    if (result.status != cutlass::Status::kSuccess) {
      std::cerr << "Failed to run CUTLASS Grouped FMHA kernel." << std::endl;
      return result;
    }

    // Wait for completion
    result.error = hipDeviceSynchronize();

    if (result.error != hipSuccess)  {
      std::cerr << "Kernel execution error: " << hipGetErrorString(result.error) << std::endl;
      return result;
    }

    //
    // Verify correctness
    //
    result.passed = true;

    if (options.reference_check) {
      result.passed = verify_();
    }

    //
    // Warm-up run of the grouped FMHA object
    //
    result.status = fmha.run();

    if (result.status != cutlass::Status::kSuccess) {
      std::cerr << "Failed to run CUTLASS Grouped FMHA kernel." << std::endl;
      return result;
    }

    //
    // Construct events
    //

    hipEvent_t events[2];

    for (auto & event : events) {
      result.error = hipEventCreate(&event);
      if (result.error != hipSuccess) {
        std::cerr << "hipEventCreate() failed: " << hipGetErrorString(result.error) << std::endl;
        return -1;
      }
    }

    // Record an event at the start of a series of FMHA operations
    result.error = hipEventRecord(events[0]);
    if (result.error != hipSuccess) {
      std::cerr << "hipEventRecord() failed: " << hipGetErrorString(result.error) << std::endl;
      return result;
    }

    //
    // Run profiling loop
    //

    for (int iter = 0; iter < this->options.iterations; ++iter) {
      fmha();
    }

    //
    // Stop profiling loop
    //

    // Record an event when the GEMM operations have been launched.
    result.error = hipEventRecord(events[1]);
    if (result.error != hipSuccess) {
      std::cerr << "hipEventRecord() failed: " << hipGetErrorString(result.error) << std::endl;
      return result;
    }

    // Wait for work on the device to complete.
    result.error = hipEventSynchronize(events[1]);
    if (result.error != hipSuccess) {
      std::cerr << "hipEventSynchronize() failed: " << hipGetErrorString(result.error) << std::endl;
      return result;
    }

    // Measure elapsed runtime
    float runtime_ms = 0;
    result.error = hipEventElapsedTime(&runtime_ms, events[0], events[1]);
    if (result.error != hipSuccess) {
      std::cerr << "cudaEventElapsed() failed: " << hipGetErrorString(result.error) << std::endl;
      return result;
    }

    // Compute average runtime and GFLOPs.
    result.runtime_ms = double(runtime_ms) / double(this->options.iterations);
    result.gflops = this->options.gflops(result.runtime_ms / 1000.0);

    //
    // Cleanup
    //

    for (auto event : events) {
      (void)hipEventDestroy(event);
    }

    std::cout << std::endl;
    // std::cout << " Attention kernel:\n"
    //   << "====================================================" << std::endl;
    // std::cout << "    " << " {seq length Q, seq length KV, head size, head size V, head number, batch size} = {" << options.seq_length \
    //   << ", " << options.seq_length_kv << ", " << options.head_size << ", " << options.head_size_v << ", " << options.head_number\
    //   << ", " << options.batch_size << "}." << std::endl;
    // options.print_problems();
    // std::cout << std::endl;
    std::cout << " Attention kernel: " << "Runtime: " << result.runtime_ms*1000 << " us" << std::endl;

    return result;
  }


};


template <
  int kQueriesPerBlock,
  int kKeysPerBlock,
  bool kSingleValueIteration,
  cutlass::gemm::kernel::GroupScheduleMode GroupScheduleMode_
>
int run_grouped(Options& options) {
  using AttentionKernel = typename cutlass::gemm::kernel::DefaultFMHAGrouped<
    cutlass::half_t,      // scalar_t
    cutlass::arch::Sm80,  // ArchTag
    true,                 // Memory is aligned
    kQueriesPerBlock,
    kKeysPerBlock,
    kSingleValueIteration,
    GroupScheduleMode_
  >::FMHAKernel;

  using FMHA = cutlass::gemm::device::GemmGrouped<AttentionKernel>;

  //
  // Test and profile
  //

  TestbedAttention<FMHA> testbed(options);

  Result result = testbed.profile();
  if (!result.passed) {
    std::cout << "Profiling CUTLASS attention has failed.\n";
    std::cout << "\nFailed\n";
    return -1;
  }

  return 0;
}


template <
  int kQueriesPerBlock,
  int kKeysPerBlock,
  bool kSingleValueIteration
>
int run_attention(Options& options) {
  if (options.scheduler_mode == cutlass::gemm::kernel::GroupScheduleMode::kDeviceOnly) {
    return run_grouped<kQueriesPerBlock,
                       kKeysPerBlock,
                       kSingleValueIteration,
                       cutlass::gemm::kernel::GroupScheduleMode::kDeviceOnly>(options);
  } else {
    return run_grouped<kQueriesPerBlock,
                       kKeysPerBlock,
                       kSingleValueIteration,
                       cutlass::gemm::kernel::GroupScheduleMode::kHostPrecompute>(options);
  }
}

void ds_kernel(Flash_fwd_params *ffp){
  

  using elem_type = cutlass::half_t;
  run_mha_fwd(*ffp,stream);

}
// =========================
// main routine
// =========================
int main (int argc, char const **args)
{
  //
  // This example uses mma.sync to directly access Tensor Cores to achieve peak performance.
  //
  
  //ds_kernel(ffp);
  //torch_launch_add2()
  //ds_kernel();
  //gemm();
  //test_type();
  //test_arr();
  //multi_add();
  //test_tensor();
  //tzqtest();
  //test_tensor_gemm();
  //test_type();


  hipDeviceProp_t props;

  hipError_t error = hipGetDeviceProperties(&props, 0);
  if (error != hipSuccess) {
    std::cerr << "hipGetDeviceProperties() returned an error: " << hipGetErrorString(error) << std::endl;
    return -1;
  }

  if (__CUDACC_VER_MAJOR__ < 11 || props.major < 8) {
  
    //
    // This example requires an NVIDIA Ampere-architecture GPU.
    //

    return 0;
  }


  //
  // Parse options
  //

  Options options;

  options.parse(argc, args);

  if (options.error) {
    std::cerr << "Aborting execution." << std::endl;
    return -1;
  }

  if (options.use_mask) {
    std::cerr << "--use_mask is not supported at the moment\n";
    return -2;
  }
  if (options.alignment != 1) {
    std::cerr << "--alignment=1 is the only supported value\n";
    return -2;
  }


  if (options.head_size_v > 64) {
    // Determine kernel configuration based on head size.
    // If head size is less than or equal to 64,
    // each block operates over 64 queries and
    // 64 keys, and partial results can be stored
    // in the register file.
    // If head size is greater than 64, 
    //each block operates over 32 queries and 128 keys,
    // and partial results are stored in shared memory.
    static int const kQueriesPerBlock = 32;
    static int const kKeysPerBlock = 128;
    if (options.head_size_v <= kKeysPerBlock) {
      return run_attention<kQueriesPerBlock, kKeysPerBlock, true>(options);
    } else {
      return run_attention<kQueriesPerBlock, kKeysPerBlock, false>(options);
    }

    hipEvent_t events[2];

    for (auto & event : events) {
      result.error = hipEventCreate(&event);
      if (result.error != hipSuccess) {
        std::cerr << "hipEventCreate() failed: " << hipGetErrorString(result.error) << std::endl;
        return -1;
      }
    }

    ds_kernel(&atten_params.ffp);

    result.error = hipEventRecord(events[0]);
    if (result.error != hipSuccess) {
      std::cerr << "hipEventRecord() failed: " << hipGetErrorString(result.error) << std::endl;
    }
    float runtime_ms = 0;
    result.error = hipEventSynchronize(events[1]);
    result.error = hipEventElapsedTime(&runtime_ms, events[0], events[1]);
      std::cout << "Runtime: " << result.runtime_ms*1000 << " us" << std::endl;

  } else {
    static int const kQueriesPerBlock = 64;
    static int const kKeysPerBlock = 64;
    return run_attention<kQueriesPerBlock, kKeysPerBlock, true>(options);
    ds_kernel(&atten_params.ffp);
    std::cout << "Runtime: " << result.runtime_ms*1000 << " us" << std::endl;

  }
  
  ds_kernel(&atten_params.ffp);

  return 0;
}